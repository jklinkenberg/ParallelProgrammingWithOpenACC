#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
******************************************************
This file is the OpenACC multi-GPU version of 2D Heat Equation 
using OpenMP+OpenACC hybrid model. This implementation is based
on the CPU version from
http://www.many-core.group.cam.ac.uk/archive/CUDAcourse09/

Permission to use, copy, distribute and modify this software for any 
purpose with or without fee is hereby granted. This software is        
provided "as is" without express or implied warranty. 

Send comments or suggestions for this OpenACC version to
            rxu6@uh.edu, schandra@udel.edu

Authors: Rengan Xu, Sunita Chandrasekaran

May 26th, 2016
******************************************************
*/

extern "C" __global__ void step_kernel(int ni, 
                     int nj,
                     double tfac, 
                     double *temp_in,
                     double *temp_out) 
{
    int i, j, i00, im10, ip10, i0m1, i0p1;
    double d2tdx2, d2tdy2;

    j = blockIdx.y + 1;
    while(j < nj-1)
    {
        i = threadIdx.x + blockIdx.x*blockDim.x + 1;
        while(i < ni-1)
        {
            i00 = i + ni*j;
            im10 = i-1 + ni*j;
            ip10 = i+1 + ni*j;
            i0m1 = i + ni*(j-1);
            i0p1 = i + ni*(j+1);

            d2tdx2 = temp_in[im10] - 2*temp_in[i00] + temp_in[ip10];
            d2tdy2 = temp_in[i0m1] - 2*temp_in[i00] + temp_in[i0p1];
            
            temp_out[i00] = temp_in[i00] + tfac*(d2tdx2 + d2tdy2);
            i += blockDim.x*gridDim.x;
        }
        j += gridDim.y;
    }
}
